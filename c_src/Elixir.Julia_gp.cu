#include "hip/hip_runtime.h"
#include "erl_nif.h"

__global__
void julia_kernel(float *ptr, int dim)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = (x + (y * dim));
	int juliaValue = 1;
	float scale = 0.1;
	float jx = ((scale * (dim - x)) / dim);
	float jy = ((scale * (dim - y)) / dim);
	float cr = (- 0.8);
	float ci = 0.156;
	float ar = jx;
	float ai = jy;
for( int i = 0; i<200; i++){
	float nar = (((ar * ar) - (ai * ai)) + cr);
	float nai = (((ai * ar) + (ar * ai)) + ci);
if((((nar * nar) + (nai * nai)) > 1000))
{
	juliaValue = 0;
break;
}

	ar = nar;
	ai = nai;
}

	ptr[((offset * 4) + 0)] = (255 * juliaValue);
	ptr[((offset * 4) + 1)] = 0;
	ptr[((offset * 4) + 2)] = 0;
	ptr[((offset * 4) + 3)] = 255;
}

extern "C" void julia_kernel_call(ErlNifEnv *env, const ERL_NIF_TERM argv[], ErlNifResourceType* type)
  {

    ERL_NIF_TERM list;
    ERL_NIF_TERM head;
    ERL_NIF_TERM tail;
    float **array_res;

    const ERL_NIF_TERM *tuple_blocks;
    const ERL_NIF_TERM *tuple_threads;
    int arity;

    if (!enif_get_tuple(env, argv[1], &arity, &tuple_blocks)) {
      printf ("spawn: blocks argument is not a tuple");
    }

    if (!enif_get_tuple(env, argv[2], &arity, &tuple_threads)) {
      printf ("spawn:threads argument is not a tuple");
    }
    int b1,b2,b3,t1,t2,t3;

    enif_get_int(env,tuple_blocks[0],&b1);
    enif_get_int(env,tuple_blocks[1],&b2);
    enif_get_int(env,tuple_blocks[2],&b3);
    enif_get_int(env,tuple_threads[0],&t1);
    enif_get_int(env,tuple_threads[1],&t2);
    enif_get_int(env,tuple_threads[2],&t3);

    dim3 blocks(b1,b2,b3);
    dim3 threads(t1,t2,t3);

    list= argv[3];

  enif_get_list_cell(env,list,&head,&tail);
  enif_get_resource(env, head, type, (void **) &array_res);
  float *arg1 = *array_res;
  list = tail;

  enif_get_list_cell(env,list,&head,&tail);
  int arg2;
  enif_get_int(env, head, &arg2);
  list = tail;

   julia_kernel<<<blocks, threads>>>(arg1,arg2);
    hipError_t error_gpu = hipGetLastError();
    if(error_gpu != hipSuccess)
     { char message[200];
       strcpy(message,"Error kernel call: ");
       strcat(message, hipGetErrorString(error_gpu));
       enif_raise_exception(env,enif_make_string(env, message, ERL_NIF_LATIN1));
     }
}
